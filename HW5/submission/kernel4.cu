#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__device__ int mandel(float c_re, 
		      float c_im, 
		      int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        float z_re_t = z_re * z_re;
	float z_im_t = z_im * z_im;
        if (z_re_t + z_im_t > 4.f)
            break;

        float new_re = z_re_t - z_im_t;
        float new_im = 2.f * z_re * z_im;
	z_re = c_re + new_re;
	z_im = c_im + new_im;
    }

    return i;
}


__global__ void mandel_kernel(int *device_img, 
			      float lower_x, 
			      float lower_y,
                              float step_x, 
			      float step_y,
                              int res_x, 
			      int res_y,
                              int max_iterations)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int this_x = blockIdx.x * blockDim.x + threadIdx.x;
    int this_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (this_x >= res_x || this_y >= res_y) 
        return;
    
    float x = lower_x + this_x * step_x;
    float y = lower_y + this_y * step_y;

    int idx = this_y * res_x + this_x;
    device_img[idx] = mandel(x, y, max_iterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void host_fe(float upper_x,
             float upper_y,
             float lower_x,
             float lower_y,
             int *img,
             int res_x,
             int res_y,
             int max_iterations)
{
    float step_x = (upper_x - lower_x) / (float)res_x;
    float step_y = (upper_y - lower_y) / (float)res_y;

    int *device_img;
    size_t size = res_x * res_y * sizeof(int);

    // Allocate device memory
    hipMalloc((void**)&device_img, size);

    // Define block and grid dimensions
    dim3 block_dim(8, 8);
    dim3 grid_dim((res_x + block_dim.x - 1) / block_dim.x, (res_y + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    mandel_kernel<<<grid_dim, block_dim>>>(device_img, lower_x, lower_y, step_x, step_y, res_x, res_y, max_iterations);

    // Ensure kernel completion
    hipDeviceSynchronize();

    // Copy result from device to image buffer
    hipMemcpy(img, device_img, size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(device_img);
}
