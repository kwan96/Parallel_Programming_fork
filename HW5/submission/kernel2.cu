#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__device__ int mandel(float c_re, 
		      float c_im, 
		      int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = (z_re * z_re) - (z_im * z_im);
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandel_kernel(int *device_img, 
			      size_t pitch,
			      float lower_x, 
			      float lower_y, 
			      float step_x, 
			      float step_y,
			      int res_x, 
			      int res_y,
			      int max_iterations)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int this_x = blockIdx.x * blockDim.x + threadIdx.x;
    int this_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (this_x >= res_x || this_y >= res_y)
        return;

    float x = lower_x + this_x * step_x;
    float y = lower_y + this_y * step_y;

    // Using pitch to index the 2D array
    int *row = (int*)((char*)device_img + this_y * pitch);
    row[this_x] = mandel(x, y, max_iterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void host_fe(float upper_x,
             float upper_y,
             float lower_x,
             float lower_y,
             int *img,
             int res_x,
             int res_y,
             int max_iterations)
{
    float step_x = (upper_x - lower_x) / (float)res_x;
    float step_y = (upper_y - lower_y) / (float)res_y;

    int *device_img;
    int *host_buffer;
    size_t pitch;

    // Allocate pinned host memory
    hipHostAlloc((void**)&host_buffer, res_x * res_y * sizeof(int), hipHostMallocDefault);

    // Allocate pitched device memory
    hipMallocPitch((void**)&device_img, &pitch, res_x * sizeof(int), res_y);

    // Set up execution configuration
    dim3 block_dim(16, 16);
    dim3 grid_dim((res_x + block_dim.x - 1) / block_dim.x, (res_y + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    mandel_kernel<<<grid_dim, block_dim>>>(device_img, pitch, lower_x, lower_y, step_x, step_y, res_x, res_y, max_iterations);

    // Ensure kernel completion
    hipDeviceSynchronize();
    
    // Copy data from pitched device memory to pinned host memory
    hipMemcpy2D(host_buffer, res_x * sizeof(int), device_img, pitch, res_x * sizeof(int), res_y, hipMemcpyDeviceToHost);

    // Copy result into the provided image buffer
    for (int i = 0; i < res_x * res_y; ++i) {
        img[i] = host_buffer[i];
    }

    // Clean up
    hipFree(device_img);
    hipHostFree(host_buffer);
}
